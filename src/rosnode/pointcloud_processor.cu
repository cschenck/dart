#include "hip/hip_runtime.h"

#include "util.h"

#include "pointcloud_processor.h"


__global__ void gpu_computeCloudMask(const ushort* depth, const int width, const int height, const float2 fl, const float2 pp, const float conversion, const float4 plane, int* mask) 
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // Run the gauntlet. Set to 0 and return if it's filtered, otherwise set to 1.
    const int index = x + y*width;
    const float z = conversion*depth[index];
    if(z > DEFAULT_DEPTH_FILTER)
    {
        mask[index] = 0;
        return;
    }

    float3 xyz = uvd_to_xyz(make_float3(x, y, z), fl, pp, width, height);
    float d = plane.x*xyz.x + plane.y*xyz.y + plane.z*xyz.z + plane.w;
    if(d < PLANE_DISTANCE_THRESHOLD)
    {
        mask[index] = 0;
        return;
    }
    
    mask[index] = 1;
}


void PointcloudProcessor::computeCloudMask(const dart::DepthSource<ushort,uchar3>* source)
{
    uint width = source->getDepthWidth();
    uint height = source->getDepthHeight();
    if(_mask.length() != width*height)
        _mask.resize(width*height);
    float2 fl = source->getFocalLength();
    float2 pp = source->getPrincipalPoint();
    const ushort* ptr = source->getDeviceDepth();
    dim3 block(16,8,1);
    dim3 grid( ceil( width / (float)block.x), ceil( height / (float)block.y ));
    
    float4 plane = _table_plane;
    plane /= sqrtf(plane.x*plane.x + plane.y*plane.y + plane.z*plane.z);

    gpu_computeCloudMask<<<grid,block>>>(ptr,width,height,fl, pp, source->getScaleToMeters(), plane, _mask.devicePtr());
}
