#include "hip/hip_runtime.h"


#include "util.h"

__device__ __host__
float3 uvd_to_xyz(float3 uvd, float2 focal_length, float2 pp, int width, int height)
{
    float3 ret;
    ret.x = (uvd.x-pp.x)*uvd.z/focal_length.x;
    ret.y = (uvd.y-pp.y)*uvd.z/focal_length.y;
    ret.z = uvd.z;
    return ret;
}

